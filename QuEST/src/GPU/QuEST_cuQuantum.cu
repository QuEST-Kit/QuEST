#include "hip/hip_runtime.h"
// Distributed under MIT licence. See https://github.com/QuEST-Kit/QuEST/blob/master/LICENCE.txt for details

/** @file
 * An implementation of QuEST's backend (../QuEST_internal.h) using NVIDIA's cuQuantum library.
 * This makes no use of the ComplexArray qureg.deviceStateVec, used by the bespoke GPU kernels,
 * which is not malloc'd in this deployment. Instead, this cuQuantum backend mallocs and uses
 * two dedicated arrays of 'cuAmp' complex primitives; qureg.cuStateVec (CPU memory) and
 * qureg.deviceCuStateVec (GPU memory)
 *
 * @author Tyson Jones
 */

# include "QuEST.h"
# include "QuEST_gpu_common.h"
# include "QuEST_precision.h"
# include "QuEST_validation.h"

# include <hip/hip_complex.h>
# include <vector>
# include <custatevec.h>
# include <thrust/device_ptr.h>



/*
 * TYPES AND ADAPTERS
 */

// precision-agnostic conversions between cuAmp, qreal and Complex
# if QuEST_PREC==1
    # define TO_CU_AMP(re, im) make_hipFloatComplex(re, im)
    # define cuAmpReal(amp) hipCrealf(amp)
    # define cuAmpImag(amp) hipCimagf(amp)
    # define cuAmpConj(amp) hipConjf(amp)
    # define CU_AMP_IN_STATE_PREC CUDA_C_F32
    # define CU_AMP_IN_MATRIX_PREC HIP_C_64F
# elif QuEST_PREC==2
    # define TO_CU_AMP(re, im) make_hipDoubleComplex(re, im)
    # define cuAmpReal(amp) hipCreal(amp)
    # define cuAmpImag(amp) hipCimag(amp)
    # define cuAmpConj(amp) hipConj(amp)
    # define CU_AMP_IN_STATE_PREC HIP_C_64F
    # define CU_AMP_IN_MATRIX_PREC HIP_C_64F
# elif QuEST_PREC==4
    # define TO_CU_AMP(re, im) -1 // invalid precision config
    # define cuAmpReal(amp) -1
    # define cuAmpImag(amp) -1
    # define CU_AMP_IN_STATE_PREC void // invalid
    # define CU_AMP_IN_MATRIX_PREC void // invalid
#endif

// convenient operator overloads for cuAmp, for doing complex artihmetic
cuAmp operator - (const cuAmp& a) {
    return TO_CU_AMP(-cuAmpReal(a), -cuAmpImag(a));
}

// convert user-facing Complex to cuQuantum-facing cuAmp
cuAmp toCuAmp(Complex c) {
    return TO_CU_AMP(c.real, c.imag);
}

// concise alias for row-wise flattened complex matrix
typedef std::vector<cuAmp> cuMatr;

// flatten ComplexMatrixN mIn to a cuMatr mOut
#define GET_cuMatr_FROM_ComplexMatrix( mOut, mIn, nQubits ) \
    long long int dim = (1LL << nQubits); \
    cuMatr mOut(dim*dim); \
    long long int i=0; \
    for (long long int r=0; r<(dim); r++) \
        for (long long int c=0; c<(dim); c++) \
            mOut[i++] = TO_CU_AMP(mIn.real[r][c], mIn.imag[r][c]);

// convert user-facing ComplexMatrixN to cuQuantum-facing cuMatr
cuMatr toCuMatr(ComplexMatrix2 mIn) {
    GET_cuMatr_FROM_ComplexMatrix(mOut, mIn, 1);
    return mOut;
}
cuMatr toCuMatr(ComplexMatrix4 mIn) {
    GET_cuMatr_FROM_ComplexMatrix(mOut, mIn, 2);
    return mOut;
}
cuMatr toCuMatr(ComplexMatrixN mIn) {
    GET_cuMatr_FROM_ComplexMatrix(mOut, mIn, mIn.numQubits);
    return mOut;
}

// convert QuEST backend masks back into user-input qubit lists (needed by cuQuantum)
std::vector<int> getIndsFromMask(long long int mask, int numBits) {
    std::vector<int> inds;
    for (int i=0; i<numBits; i++)
        if (mask & (1LL<<i))
            inds.push_back(i);
    return inds;
}



#ifdef __cplusplus
extern "C" {
#endif



/*
 * CUQUANTUM WRAPPERS (to reduce boilerplate)
 */

void custatevec_applyMatrix(Qureg qureg, std::vector<int> ctrls, std::vector<int> targs, cuMatr matr) {

    // do not adjoint matrix
    int adj = 0;

    // condition all ctrls on =1 state
    int* ctrlBits = nullptr;

    // use automatic workspace management
    void* work = nullptr;
    size_t workSize = 0;

    custatevecApplyMatrix(
        qureg.cuQuantumHandle, 
        qureg.deviceCuStateVec, CU_AMP_IN_STATE_PREC, qureg.numQubitsInStateVec, 
        matr.data(), CU_AMP_IN_MATRIX_PREC, CUSTATEVEC_MATRIX_LAYOUT_ROW, adj, 
        targs.data(), targs.size(), 
        ctrls.data(), ctrlBits, ctrls.size(), 
        CUSTATEVEC_COMPUTE_DEFAULT,
        work, workSize);
}



/* 
 * ENVIRONMENT MANAGEMENT
 */

int GPUSupportsMemPools() {

    // consult only the first device (garuanteed already to exist)
    int device = 0;

    int supports;
    hipDeviceGetAttribute(&supports, hipDeviceAttributeMemoryPoolsSupported, device);
    return supports;
}

int memPoolAlloc(void* ctx, void** ptr, size_t size, hipStream_t stream) {
    hipMemPool_t pool = *static_cast<hipMemPool_t*>(ctx);
    return hipMallocFromPoolAsync(ptr, size, pool, stream); 
}
int memPoolFree(void* ctx, void* ptr, size_t size, hipStream_t stream) {
    return hipFreeAsync(ptr, stream); 
}

void setupAutoWorkspaces(custatevecHandle_t cuQuantumHandle) {

    // get the current (device's default) stream-ordered memory pool (assuming single GPU)
    int deviceId;
    hipGetDevice(&deviceId);
    hipMemPool_t memPool;
    hipDeviceGetMemPool(&memPool, deviceId);

    // get its current memory threshold, above which memory gets freed at every stream synch
    size_t currMaxMem;
    hipMemPoolGetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &currMaxMem); 

    // if it's smaller than 1 MiB = 16 qubits, extend it
    size_t desiredMaxMem = 16*(1<<15);
    if (currMaxMem < desiredMaxMem)
        hipMemPoolSetAttribute(memPool, hipMemPoolAttrReleaseThreshold, &desiredMaxMem); 

    // create a mem handler around the mem pool
    custatevecDeviceMemHandler_t memHandler;
    memHandler.ctx = &memPool;
    memHandler.device_alloc = memPoolAlloc;
    memHandler.device_free = memPoolFree;
    strcpy(memHandler.name, "mempool");
    printf("created mem handler\n");

    // set cuQuantum to use this handler and pool, to automate workspace memory management
    custatevecSetDeviceMemHandler(cuQuantumHandle, &memHandler);
}

QuESTEnv createQuESTEnv(void) {
    validateGPUExists(GPUExists(), __func__);
    validateGPUIsCuQuantumCompatible(GPUSupportsMemPools(),__func__);
    
    QuESTEnv env;
    env.rank=0;
    env.numRanks=1;
    
    env.seeds = NULL;
    env.numSeeds = 0;
    seedQuESTDefault(&env);

    // prepare cuQuantum
    custatevecCreate(&env.cuQuantumHandle);
    hipStreamCreate(&env.cuStream);    
    custatevecSetStream(env.cuQuantumHandle, env.cuStream);
    setupAutoWorkspaces(env.cuQuantumHandle);
    
    return env;
}

void destroyQuESTEnv(QuESTEnv env){
    free(env.seeds);

    // finalise cuQuantum
    custatevecDestroy(env.cuQuantumHandle);
    hipStreamDestroy(env.cuStream);
}



/* 
 * QUREG CREATION AND AMP SET/GET
 */

void statevec_createQureg(Qureg *qureg, int numQubits, QuESTEnv env)
{   
    // set standard fields
    long long int numAmps = 1LL << numQubits;
    qureg->numQubitsInStateVec = numQubits;
    qureg->numAmpsPerChunk = numAmps;
    qureg->numAmpsTotal = numAmps;
    qureg->chunkId = 0;
    qureg->numChunks = 1;
    qureg->isDensityMatrix = 0;

    // copy env's cuQuantum handles
    qureg->cuQuantumHandle = env.cuQuantumHandle;
    qureg->cuStream = env.cuStream;

    // allocate user-facing CPU memory
    qureg->stateVec.real = (qreal*) malloc(numAmps * sizeof(qureg->stateVec.real));
    qureg->stateVec.imag = (qreal*) malloc(numAmps * sizeof(qureg->stateVec.imag));
    validateQuregAllocation(qureg, env, __func__);

    // allocate cuQuantum GPU memory (unvalidated)
    hipMalloc( &(qureg->deviceCuStateVec), numAmps * sizeof(*(qureg->deviceCuStateVec)) );

    // allocate private cuQuantum CPU memory (for exchanging with GPU memory)
    qureg->cuStateVec = (cuAmp*) malloc(numAmps * sizeof(*(qureg->cuStateVec)));
}

void statevec_destroyQureg(Qureg qureg, QuESTEnv env)
{
    // free user-facing CPU memory 
    free(qureg.stateVec.real);
    free(qureg.stateVec.imag);
    
    // free private cuQuantum CPU memory
    free(qureg.cuStateVec);

    // free cuQuantum GPU memory
    hipFree(qureg.deviceCuStateVec);
}

void statevec_setAmps(Qureg qureg, long long int startInd, qreal* reals, qreal* imags, long long int numAmps)
{
    // slowly manually overwrite subset of private cuQuantum CPU memory
    for (long long int i=0; i<numAmps; i++)
        qureg.cuStateVec[i+startInd] = TO_CU_AMP(reals[i], imags[i]);

    // cuda-copy subset to GPU memory subset
    hipDeviceSynchronize();
    hipMemcpy(
        &(qureg.deviceCuStateVec[startInd]), 
        &(qureg.cuStateVec[startInd]), 
        numAmps * sizeof(cuAmp), hipMemcpyHostToDevice);
}

void statevec_copySubstateToGPU(Qureg qureg, long long int startInd, long long int numAmps)
{
    statevec_setAmps(qureg, startInd, &(qureg.stateVec.real[startInd]), &(qureg.stateVec.imag[startInd]), numAmps);
}

void statevec_copySubstateFromGPU(Qureg qureg, long long int startInd, long long int numAmps)
{
    // cuda-copy subset of GPU memory to private cuQuantum CPU memory
    hipDeviceSynchronize();
    hipMemcpy(
        &(qureg.cuStateVec[startInd]), 
        &(qureg.deviceCuStateVec[startInd]), 
        numAmps * sizeof(*(qureg.cuStateVec)), 
        hipMemcpyDeviceToHost);

    // slowly manually overwrite public CPU memory from private
    for (long long int i=startInd; i<(startInd+numAmps); i++) {
        qureg.stateVec.real[i] = cuAmpReal(qureg.cuStateVec[i]);
        qureg.stateVec.imag[i] = cuAmpImag(qureg.cuStateVec[i]);
    }
}

void copyStateToGPU(Qureg qureg)
{
    statevec_copySubstateToGPU(qureg, 0, qureg.numAmpsTotal);
}

void copyStateFromGPU(Qureg qureg)
{
    statevec_copySubstateFromGPU(qureg, 0, qureg.numAmpsTotal);
}

void statevec_cloneQureg(Qureg targetQureg, Qureg copyQureg)
{
    // directly cuda-copy the GPU memory 
    hipDeviceSynchronize();
    hipMemcpy(
        targetQureg.deviceCuStateVec,
        copyQureg.deviceCuStateVec,
        copyQureg.numAmpsTotal * sizeof(cuAmp),
        hipMemcpyDeviceToDevice);
}

qreal statevec_getRealAmp(Qureg qureg, long long int index)
{
    cuAmp amp;
    hipDeviceSynchronize();
    hipMemcpy(&amp, &(qureg.deviceCuStateVec[index]), sizeof(cuAmp), hipMemcpyDeviceToHost);
    return cuAmpReal(amp);
}

qreal statevec_getImagAmp(Qureg qureg, long long int index)
{
    cuAmp amp;
    hipDeviceSynchronize();
    hipMemcpy(&amp, &(qureg.deviceCuStateVec[index]), sizeof(cuAmp), hipMemcpyDeviceToHost);
    return cuAmpImag(amp);
}



/*
 * STATE INITIALISATION
 */

void densmatr_initPureState(Qureg targetQureg, Qureg copyQureg)
{
}

void densmatr_initPlusState(Qureg qureg)
{
}

void densmatr_initClassicalState(Qureg qureg, long long int stateInd)
{
}

void statevec_initBlankState(Qureg qureg)
{
}

void statevec_initZeroState(Qureg qureg)
{
}

void statevec_initPlusState(Qureg qureg)
{
}

void statevec_initClassicalState(Qureg qureg, long long int stateInd)
{
}

void statevec_initDebugState(Qureg qureg)
{
}

void statevec_initStateOfSingleQubit(Qureg *qureg, int qubitId, int outcome)
{
}

int statevec_initStateFromSingleFile(Qureg *qureg, char filename[200], QuESTEnv env)
{
    return -1;
}

void densmatr_setQuregToPauliHamil(Qureg qureg, PauliHamil hamil)
{
}

void statevec_setWeightedQureg(Complex fac1, Qureg qureg1, Complex fac2, Qureg qureg2, Complex facOut, Qureg out)
{
}



/*
 * DEBUG
 */

void statevec_reportStateToScreen(Qureg qureg, QuESTEnv env, int reportRank)
{
}

int statevec_compareStates(Qureg mq1, Qureg mq2, qreal precision)
{
    return -1;
}



/*
 * OPERATORS
 */

void statevec_compactUnitary(Qureg qureg, int targetQubit, Complex alpha, Complex beta) 
{
    cuAmp a = toCuAmp(alpha);
    cuAmp b = toCuAmp(beta);
    cuMatr matrix{
        a, -cuAmpConj(b),
        b,  cuAmpConj(a)
    };
    custatevec_applyMatrix(qureg, {}, {targetQubit}, matrix);
}

void statevec_controlledCompactUnitary(Qureg qureg, int controlQubit, int targetQubit, Complex alpha, Complex beta) 
{
    cuAmp a = toCuAmp(alpha);
    cuAmp b = toCuAmp(beta);
    cuMatr matrix{
        a, -cuAmpConj(b),
        b,  cuAmpConj(a)
    };
    custatevec_applyMatrix(qureg, {controlQubit}, {targetQubit}, matrix);
}

void statevec_unitary(Qureg qureg, int targetQubit, ComplexMatrix2 u)
{
    custatevec_applyMatrix(qureg, {}, {targetQubit}, toCuMatr(u));
}

void statevec_multiControlledMultiQubitUnitary(Qureg qureg, long long int ctrlMask, int* targs, int numTargs, ComplexMatrixN u)
{
    std::vector<int> c = getIndsFromMask(ctrlMask,qureg.numQubitsInStateVec);
    std::vector<int> t(targs,targs+numTargs); 
    custatevec_applyMatrix(qureg, c, t, toCuMatr(u));
}

void statevec_multiControlledTwoQubitUnitary(Qureg qureg, long long int ctrlMask, int q1, int q2, ComplexMatrix4 u)
{
    std::vector<int> c = getIndsFromMask(ctrlMask,qureg.numQubitsInStateVec);
    custatevec_applyMatrix(qureg, c, {q1,q2}, toCuMatr(u));
}

void statevec_controlledUnitary(Qureg qureg, int controlQubit, int targetQubit, ComplexMatrix2 u)
{
    custatevec_applyMatrix(qureg, {controlQubit}, {targetQubit}, toCuMatr(u));
}

void statevec_multiControlledUnitary(Qureg qureg, long long int ctrlQubitsMask, long long int ctrlFlipMask, int targetQubit, ComplexMatrix2 u)
{
    int targs[] = {targetQubit};
    std::vector<int> ctrlInds = getIndsFromMask(ctrlQubitsMask,qureg.numQubitsInStateVec);
    std::vector<int> ctrlVals(ctrlInds.size());
    for (size_t i=0; i<ctrlInds.size(); i++)
        ctrlVals[i] = !(ctrlFlipMask & (1LL<<ctrlInds[i]));

    custatevecApplyMatrix(
        qureg.cuQuantumHandle, 
        qureg.deviceCuStateVec, CU_AMP_IN_STATE_PREC, qureg.numQubitsInStateVec, 
        toCuMatr(u).data(), CU_AMP_IN_MATRIX_PREC, CUSTATEVEC_MATRIX_LAYOUT_ROW, 0, 
        targs, 1, ctrlInds.data(), ctrlVals.data(), ctrlInds.size(), 
        CUSTATEVEC_COMPUTE_DEFAULT, nullptr, 0);
}

void statevec_pauliX(Qureg qureg, int targetQubit) 
{
    cuAmp a0 = TO_CU_AMP(0, 0);
    cuAmp a1 = TO_CU_AMP(1, 0);
    cuMatr matrix{
        a0, a1,
        a1, a0
    };
    custatevec_applyMatrix(qureg, {}, {targetQubit}, matrix);
}

void statevec_pauliY(Qureg qureg, int targetQubit) 
{
    cuAmp a0 = TO_CU_AMP(0, 0);
    cuAmp aI = TO_CU_AMP(0, 1);
    cuMatr matrix{
        a0, -aI,
        aI,  a0
    };
    custatevec_applyMatrix(qureg, {}, {targetQubit}, matrix);
}

void statevec_pauliYConj(Qureg qureg, int targetQubit) 
{
    cuAmp a0 = TO_CU_AMP(0, 0);
    cuAmp aI = TO_CU_AMP(0, 1);
    cuMatr matrix{
         a0, aI,
        -aI, a0
    };
    custatevec_applyMatrix(qureg, {}, {targetQubit}, matrix);
}

void statevec_controlledPauliY(Qureg qureg, int controlQubit, int targetQubit)
{
    cuAmp a0 = TO_CU_AMP(0, 0);
    cuAmp aI = TO_CU_AMP(0, 1);
    cuMatr matrix{
        a0, -aI,
        aI,  a0
    };
    custatevec_applyMatrix(qureg, {controlQubit}, {targetQubit}, matrix);
}

void statevec_controlledPauliYConj(Qureg qureg, int controlQubit, int targetQubit)
{
    cuAmp a0 = TO_CU_AMP(0, 0);
    cuAmp aI = TO_CU_AMP(0, 1);
    cuMatr matrix{
         a0, aI,
        -aI, a0
    };
    custatevec_applyMatrix(qureg, {controlQubit}, {targetQubit}, matrix);
}

void statevec_phaseShiftByTerm(Qureg qureg, int targetQubit, Complex term)
{   
    // this diagonal operator, otherwise embarrasingly parallel with unit stride, 
    // is here treated as a generic one-target unitary, wastefully inducing non-unit 
    // stride and unnecessary memory reads, and potentially unnecessary communication
    // in multi-GPU mode. 

    cuAmp a0 = TO_CU_AMP(0, 0);
    cuAmp a1 = TO_CU_AMP(1, 0);
    cuAmp aE = toCuAmp(term);
    cuMatr matrix{
        a1, a0,
        a0, aE
    };
    custatevec_applyMatrix(qureg, {}, {targetQubit}, matrix);
}

void statevec_controlledPhaseShift(Qureg qureg, int idQubit1, int idQubit2, qreal angle)
{
    // this diagonal operator, otherwise embarrasingly parallel with unit stride, 
    // is here treated as a generic one-target unitary, wastefully inducing non-unit 
    // stride and unnecessary memory reads, and potentially unnecessary communication
    // in multi-GPU mode. 

    cuAmp a0 = TO_CU_AMP(0, 0);
    cuAmp a1 = TO_CU_AMP(1, 0);
    cuAmp aE = TO_CU_AMP(cos(angle), sin(angle));
    cuMatr matrix{
        a1, a0,
        a0, aE
    };
    custatevec_applyMatrix(qureg, {idQubit1}, {idQubit2}, matrix);
}

void statevec_multiControlledPhaseShift(Qureg qureg, int *controlQubits, int numControlQubits, qreal angle)
{   
    // this diagonal operator, otherwise embarrasingly parallel with unit stride, 
    // is here treated as a generic one-target unitary, wastefully inducing non-unit 
    // stride and unnecessary memory reads, and potentially unnecessary communication
    // in multi-GPU mode. 

    cuAmp a0 = TO_CU_AMP(0, 0);
    cuAmp a1 = TO_CU_AMP(1, 0);
    cuAmp aE = TO_CU_AMP(cos(angle), sin(angle));
    cuMatr matrix{
        a1, a0,
        a0, aE
    };
    std::vector<int> targs{controlQubits[0]};
    std::vector<int> ctrls(controlQubits + 1, controlQubits + numControlQubits); 
    custatevec_applyMatrix(qureg, ctrls, targs, matrix);
}

void statevec_multiRotateZ(Qureg qureg, long long int mask, qreal angle)
{   
    qreal theta = - angle/2.;
    std::vector<int> targs = getIndsFromMask(mask, qureg.numQubitsInStateVec);
    std::vector<custatevecPauli_t> paulis(targs.size(), CUSTATEVEC_PAULI_Z);

    custatevecApplyPauliRotation(
        qureg.cuQuantumHandle, qureg.deviceCuStateVec, 
        CU_AMP_IN_STATE_PREC, qureg.numQubitsInStateVec, 
        theta, paulis.data(), targs.data(), targs.size(),
        nullptr, nullptr, 0);
}

void statevec_multiControlledMultiRotateZ(Qureg qureg, long long int ctrlMask, long long int targMask, qreal angle)
{   
    qreal theta = - angle/2.;
    std::vector<int> ctrls = getIndsFromMask(ctrlMask, qureg.numQubitsInStateVec);
    std::vector<int> targs = getIndsFromMask(targMask, qureg.numQubitsInStateVec);
    std::vector<custatevecPauli_t> paulis(targs.size(), CUSTATEVEC_PAULI_Z);

    custatevecApplyPauliRotation(
        qureg.cuQuantumHandle, qureg.deviceCuStateVec, 
        CU_AMP_IN_STATE_PREC, qureg.numQubitsInStateVec, 
        theta, paulis.data(), targs.data(), targs.size(),
        ctrls.data(), nullptr, ctrls.size());
}

void statevec_controlledPhaseFlip(Qureg qureg, int idQubit1, int idQubit2)
{
    // this diagonal operator, otherwise embarrasingly parallel with unit stride, 
    // is here treated as a generic one-target unitary, wastefully inducing non-unit 
    // stride and unnecessary memory reads, and potentially unnecessary communication
    // in multi-GPU mode. 

    cuAmp a0 = TO_CU_AMP(0, 0);
    cuAmp a1 = TO_CU_AMP(1, 0);
    cuMatr matrix{
        a1,  a0,
        a0, -a1
    };
    custatevec_applyMatrix(qureg, {idQubit1}, {idQubit2}, matrix);
}

void statevec_multiControlledPhaseFlip(Qureg qureg, int *controlQubits, int numControlQubits)
{
    // this diagonal operator, otherwise embarrasingly parallel with unit stride, 
    // is here treated as a generic one-target unitary, wastefully inducing non-unit 
    // stride and unnecessary memory reads, and potentially unnecessary communication
    // in multi-GPU mode. 

    cuAmp a0 = TO_CU_AMP(0, 0);
    cuAmp a1 = TO_CU_AMP(1, 0);
    cuMatr matrix{
        a1,  a0,
        a0, -a1
    };
    std::vector<int> targs{controlQubits[0]};
    std::vector<int> ctrls(controlQubits + 1, controlQubits + numControlQubits); 
    custatevec_applyMatrix(qureg, ctrls, targs, matrix);
}

void statevec_swapQubitAmps(Qureg qureg, int qb1, int qb2) 
{
    int2 targPairs[] = {{qb1, qb2}}; 
    int numPairs = 1;

    custatevecSwapIndexBits(
        qureg.cuQuantumHandle, qureg.deviceCuStateVec, 
        CU_AMP_IN_STATE_PREC, qureg.numQubitsInStateVec, 
        targPairs, numPairs,
        nullptr, nullptr, 0);
}

void statevec_hadamard(Qureg qureg, int targetQubit) 
{
    cuAmp a = TO_CU_AMP(1/sqrt(2.), 0);
    cuMatr matrix{
        a,  a,
        a, -a
    };
    custatevec_applyMatrix(qureg, {}, {targetQubit}, matrix);
}

void statevec_controlledNot(Qureg qureg, int controlQubit, int targetQubit)
{
    cuAmp a0 = TO_CU_AMP(0, 0);
    cuAmp a1 = TO_CU_AMP(1, 0);
    cuMatr matrix{
        a0, a1,
        a1, a0
    };
    custatevec_applyMatrix(qureg, {controlQubit}, {targetQubit}, matrix);
}

void statevec_multiControlledMultiQubitNot(Qureg qureg, int ctrlMask, int targMask)
{
    // this operator can be effected in one-shot using a custom kernel, but we here
    // isntead resort to slowly (by at most a factor #targs) effect it as a sequence
    // of one-target multi-ctrl NOT gates.

    cuAmp a0 = TO_CU_AMP(0, 0);
    cuAmp a1 = TO_CU_AMP(1, 0);
    cuMatr matrix{
        a0, a1,
        a1, a0
    };
    std::vector<int> ctrls = getIndsFromMask(ctrlMask, qureg.numQubitsInStateVec);
    std::vector<int> targs = getIndsFromMask(targMask, qureg.numQubitsInStateVec);
    for (int targ : targs)
        custatevec_applyMatrix(qureg, ctrls, {targ}, matrix);
}

void statevec_applySubDiagonalOp(Qureg qureg, int* targets, SubDiagonalOp op, int conj)
{
    // sneakily leverage the CPU cuQuantum memory in order to convert op
    // (as separate arrays op.real and op.imag) into cuAmp*
    cuAmp* diagonals = qureg.cuStateVec;
    for (long long int i=0; i<op.numElems; i++)
        diagonals[i] = TO_CU_AMP(op.real[i], op.imag[i]);

    custatevecApplyGeneralizedPermutationMatrix(
        qureg.cuQuantumHandle, qureg.deviceCuStateVec,
        CU_AMP_IN_STATE_PREC, qureg.numQubitsInStateVec,
        nullptr, diagonals, CU_AMP_IN_MATRIX_PREC, 0, 
        targets, op.numQubits, nullptr, nullptr, 0,
        nullptr, 0);
}

void statevec_applyDiagonalOp(Qureg qureg, DiagonalOp op) 
{
}

void densmatr_applyDiagonalOp(Qureg qureg, DiagonalOp op)
{
}

void statevec_applyPhaseFuncOverrides(
    Qureg qureg, int* qubits, int numQubits, enum bitEncoding encoding,
    qreal* coeffs, qreal* exponents, int numTerms, 
    long long int* overrideInds, qreal* overridePhases, int numOverrides,
    int conj)
{
}

void statevec_applyMultiVarPhaseFuncOverrides(
    Qureg qureg, int* qubits, int* numQubitsPerReg, int numRegs, enum bitEncoding encoding,
    qreal* coeffs, qreal* exponents, int* numTermsPerReg, 
    long long int* overrideInds, qreal* overridePhases, int numOverrides,
    int conj)
{
}

void statevec_applyParamNamedPhaseFuncOverrides(
    Qureg qureg, int* qubits, int* numQubitsPerReg, int numRegs, enum bitEncoding encoding,
    enum phaseFunc phaseFuncName, qreal* params, int numParams,
    long long int* overrideInds, qreal* overridePhases, int numOverrides,
    int conj)
{
}



/*
 * DECOHERENCE
 */

void densmatr_mixDensityMatrix(Qureg combineQureg, qreal otherProb, Qureg otherQureg)
{
}

void densmatr_mixDephasing(Qureg qureg, int targetQubit, qreal dephase) 
{
}

void densmatr_mixTwoQubitDephasing(Qureg qureg, int qubit1, int qubit2, qreal dephase)
{
}

void densmatr_mixDepolarising(Qureg qureg, int targetQubit, qreal depolLevel)
{
}

void densmatr_mixDamping(Qureg qureg, int targetQubit, qreal damping)
{
}

void densmatr_mixTwoQubitDepolarising(Qureg qureg, int qubit1, int qubit2, qreal depolLevel)
{
}



/*
 * CALCULATIONS
 */

qreal densmatr_calcTotalProb(Qureg qureg)
{
    return -1;
}

qreal statevec_calcTotalProb(Qureg qureg)
{
    return -1;
}

qreal statevec_calcProbOfOutcome(Qureg qureg, int measureQubit, int outcome)
{
    return -1;
}

qreal densmatr_calcProbOfOutcome(Qureg qureg, int measureQubit, int outcome)
{
    return -1;
}

void statevec_calcProbOfAllOutcomes(qreal* outcomeProbs, Qureg qureg, int* qubits, int numQubits)
{
}

void densmatr_calcProbOfAllOutcomes(qreal* outcomeProbs, Qureg qureg, int* qubits, int numQubits)
{
}

qreal densmatr_calcInnerProduct(Qureg a, Qureg b)
{
    return -1;
}

Complex statevec_calcInnerProduct(Qureg bra, Qureg ket)
{
    return (Complex) {.real=-1, .imag=-1};
}

qreal densmatr_calcFidelity(Qureg qureg, Qureg pureState)
{
    return -1;
}

qreal densmatr_calcHilbertSchmidtDistance(Qureg a, Qureg b)
{
    return -1;
}

qreal densmatr_calcPurity(Qureg qureg)
{
    return -1;
}

Complex statevec_calcExpecDiagonalOp(Qureg qureg, DiagonalOp op)
{
    return (Complex) {.real=-1, .imag=-1};
}

Complex densmatr_calcExpecDiagonalOp(Qureg qureg, DiagonalOp op)
{
    return (Complex) {.real=-1, .imag=-1};
}



/*
 * REDUCTIONS
 */

void statevec_collapseToKnownProbOutcome(Qureg qureg, int measureQubit, int outcome, qreal outcomeProb)
{        
}

void densmatr_collapseToKnownProbOutcome(Qureg qureg, int measureQubit, int outcome, qreal outcomeProb)
{
}



#ifdef __cplusplus
}
#endif
